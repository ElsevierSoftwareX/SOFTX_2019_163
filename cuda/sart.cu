#include "hip/hip_runtime.h"
#include "hemi/parallel_for.h"
#include <cmath>

#include "device_data.hpp"

#include "fmt/format.h"

namespace tomo {
namespace cuda {

#define EPSILON 1e-6

template <typename T>
__global__ void w_norms_kernel(const device_line<T>* device_lines,
                               device_volume v, T* w_norms) {
    // gpu, first just 'closest' kernel (value 1)
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    T result = 0;
    vec2<T> current = device_lines[i].origin;
    while (current.x <= v.x + EPSILON && current.y <= v.y + EPSILON &&
           current.x + EPSILON >= 0 && current.y + EPSILON >= 0) {
        int index = (int)(current.x) + v.x * (int)(current.y);
        if (index < v.x * v.y && index >= 0) {
            result += (T)1.0;
        }

        current.x += device_lines[i].delta.x;
        current.y += device_lines[i].delta.y;
    }

    w_norms[i] = result;
}

template <typename T>
__global__ void sart_kernel(T* device_image, const device_line<T>* device_lines,
                            const T* device_sino, device_volume v, T beta,
                            const T* w_norms) {
    // gpu, first just 'closest' kernel (value 1)
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (w_norms[i] < EPSILON)
        return;

    T alpha = 0;
    vec2<T> current = device_lines[i].origin;
    while (current.x <= v.x + EPSILON && current.y <= v.y + EPSILON &&
           current.x + EPSILON >= 0 && current.y + EPSILON >= 0) {
        int index = (int)(current.x) + v.x * (int)(current.y);
        if (index < v.x * v.y && index >= 0) {
            alpha += device_image[index];
        }

        current.x += device_lines[i].delta.x;
        current.y += device_lines[i].delta.y;
    }

    auto factor = beta * (device_sino[i] - alpha) / w_norms[i];

    current = device_lines[i].origin;
    while (current.x <= v.x + EPSILON && current.y <= v.y + EPSILON &&
           current.x + EPSILON >= 0 && current.y + EPSILON >= 0) {
        int index = (int)(current.x) + v.x * (int)(current.y);
        if (index < v.x * v.y && index >= 0) {
            device_image[index] += factor;
        }

        current.x += device_lines[i].delta.x;
        current.y += device_lines[i].delta.y;
    }
}

template <typename T>
void run_sart(device_volume v, device_line<T>* device_lines, int lines,
              T* device_sino, T* host_image, int group_count, T beta = 0.5,
              int iterations = 10) {
    T* device_image = nullptr;
    auto image_bytes = v.x * v.y * sizeof(T);

    hipMalloc(&device_image, image_bytes);
    hipMemset(device_image, 0, image_bytes);

    int group_size = lines / group_count;
    int threads = 256;

    T* w_norms = nullptr;
    auto w_norms_bytes = lines * sizeof(T);
    hipMalloc(&w_norms, w_norms_bytes);
    hipMemset(w_norms, 0, w_norms_bytes);
    w_norms_kernel<<<lines / threads, threads>>>(device_lines, v, w_norms);

    hipDeviceSynchronize();
    std::vector<T> w(lines);
    hipMemcpy(w.data(), w_norms, w_norms_bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i < iterations; ++i) {
        for (int k = 0; k < group_count; ++k) {
            sart_kernel<<<group_size / threads, threads>>>(
                device_image, &device_lines[k * group_size],
                &device_sino[k * group_size], v, beta,
                &w_norms[k * group_size]);
        }
    }

    hipMemcpy(host_image, device_image, image_bytes, hipMemcpyDeviceToHost);

    hipFree(w_norms);
    hipFree(device_image);
}

template void run_sart(device_volume v, device_line<float>* device_lines,
                       int lines, float* device_sino, float* host_image, int,
                       float beta, int iterations);

template void run_sart(device_volume v, device_line<double>* device_lines,
                       int lines, double* device_sino, double* host_image,
                       int group_count, double beta, int iterations);

} // namespace cuda
} // namespace tomo
