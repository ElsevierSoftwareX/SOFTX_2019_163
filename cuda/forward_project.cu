#include "hip/hip_runtime.h"
#include "fmt/format.h"
#include "hemi/parallel_for.h"

#include "device_data.hpp"
#include "projectors/closest.cu.hpp"

namespace tomo {
namespace cuda {

template <typename T>
__global__ void forward_project_kernel(T* device_image,
                                       device::line<T>* device_lines,
                                       T* device_sino, device::volume v) {
    // gpu, first just 'closest' kernel (value 1)
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    T result = 0;
    project_closest(device_lines[i], v, [&result, &device_image](int index) {
        result += device_image[index];
    });
    device_sino[i] = result;
}

template <typename T>
void run_forward_projection(T* device_image, device::line<T>* device_lines,
                            int lines, T* host_sino, device::volume v) {
    T* device_sino = nullptr;
    auto sino_bytes = lines * sizeof(T);
    hipMalloc(&device_sino, sino_bytes);
    hipMemset(device_sino, 0, sino_bytes);

    int threads = 256;
    forward_project_kernel<<<lines / threads, threads>>>(
        device_image, device_lines, device_sino, v);

    hipMemcpy(host_sino, device_sino, sino_bytes, hipMemcpyDeviceToHost);

    hipFree(device_sino);
}

template void run_forward_projection<float>(float*, device::line<float>*, int,
                                            float*, device::volume);

template void run_forward_projection<double>(double*, device::line<double>*, int,
                                             double*, device::volume);

} // namespace cuda
} // namespace tomo
